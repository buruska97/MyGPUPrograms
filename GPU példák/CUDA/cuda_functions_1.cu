#include "hip/hip_runtime.h"
﻿#include "cuda_functions_1.h"

__global__ void mapEvenNums(int* d_input, int* d_output, const size_t dataSize);
__global__ void scan_upkernel(int* data, const size_t dataSize, int step);
__global__ void scan_downkernel(int* data, const size_t dataSize, int step);
__global__ void copy(int* d_input, int* d_output, const size_t dataSize);
__global__ void compact_final(int* d_input, int* d_output, const size_t dataSize, int* d_mapresult, int* d_scanresult);

__host__ void scan(int* d_input, int* d_output, const size_t dataSize, dim3 dimBlock, dim3 dimGrid);

//Host function
__host__ int* cudaCompactEven(int* data, const size_t dataSize, size_t* outSize, dim3 dimBlock, dim3 dimGrid)
{
	int* d_input = nullptr;
	int* d_mapresult = nullptr;
	int* d_scanresult = nullptr;
	int* d_finalresult = nullptr;

	hipMalloc(&d_input, sizeof(int) * dataSize);
	hipMalloc(&d_mapresult, sizeof(int) * dataSize);
	hipMalloc(&d_scanresult, sizeof(int) * dataSize);
	hipMalloc(&d_finalresult, sizeof(int) * dataSize);

	hipMemcpy(d_input, data, sizeof(int) * dataSize, hipMemcpyHostToDevice);


	mapEvenNums<<<dimGrid, dimBlock>>> (d_input, d_mapresult, dataSize);
	scan(d_mapresult, d_scanresult, dataSize, dimBlock, dimGrid);
	compact_final<<<dimGrid, dimBlock>>>(d_input, d_finalresult, dataSize, d_mapresult, d_scanresult);

	
	int* h_scanresult = (int*)malloc(sizeof(int) * dataSize);
	hipMemcpy(h_scanresult, d_scanresult, sizeof(int) * dataSize, hipMemcpyDeviceToHost);

	size_t resultSize = h_scanresult[dataSize -1];

	int* h_result = (int*)malloc(sizeof(int) * resultSize);
	hipMemcpy(h_result, d_finalresult, sizeof(int) * resultSize, hipMemcpyDeviceToHost);

	hipFree(d_input);
	hipFree(d_mapresult);
	hipFree(d_scanresult);
	hipFree(d_finalresult);
	free(h_scanresult);

	*outSize = resultSize;

	return h_result;
}

__host__ void scan(int* d_input, int* d_output, const size_t dataSize, dim3 dimBlock, dim3 dimGrid)
{
	copy<<<dimGrid, dimBlock>>>(d_input, d_output, dataSize);
	int step;
	for (step = 1; step < dataSize; step *= 2)
	{
		scan_upkernel<<<dimGrid, dimBlock>>>(d_output, dataSize, step);
	}
	step /= 2; //Biggest viable step
	for (step; step > 0; step /= 2)
	{
		scan_downkernel<<<dimGrid, dimBlock>>>(d_output, dataSize, step);
	}
}


//Device Map function
__global__ void mapEvenNums(int* d_input, int* d_output, const size_t dataSize)
{
	
	for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < dataSize; idx += blockDim.x * gridDim.x)
	{
		if (idx < dataSize)
		{
			d_output[idx] = d_input[idx] * d_input[idx];

			if (d_input[idx] % 2 == 0)
				d_output[idx] = 1;
			else
				d_output[idx] = 0; 
		}
	}
	__syncthreads();
}

__global__ void copy(int* d_input, int* d_output, const size_t dataSize)
{
	for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < dataSize; id += blockDim.x * gridDim.x)
	{
		d_output[id] = d_input[id];
	}
	__syncthreads();
}

__global__ void scan_upkernel(int* data, const size_t dataSize, int step)
{
	for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < dataSize; id += blockDim.x * gridDim.x)
	{
		int sumIndex = dataSize - 1 - id * step * 2;

		if (sumIndex - step >= 0)
			data[sumIndex] += data[sumIndex - step];
	}
	__syncthreads();
}

__global__ void scan_downkernel(int* data, const size_t dataSize, int step)
{
	for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < dataSize; id += blockDim.x * gridDim.x)
	{
		int sumIndex = dataSize - 1 - (id + 1) * step * 2 + step;

		if (sumIndex - step >= 0)
			data[sumIndex] += data[sumIndex - step];
	}
	__syncthreads();
}

__global__ void compact_final(int* d_input, int* d_output, const size_t dataSize, int* d_mapresult, int* d_scanresult)
{
	for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < dataSize; id += blockDim.x * gridDim.x)
	{
		if (d_mapresult[id] == 1)
		{
			d_output[d_scanresult[id] - 1] = d_input[id];
		}
	}
	__syncthreads();
}